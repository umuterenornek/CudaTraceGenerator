#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>

#define PBSTR "||||||||||||||||||||||||||||||||||||||||||||||||||||||||||||"
#define PBWIDTH 60

void printProgress(double percentage) {
    int val = (int)(percentage * 100);
    int lpad = (int)(percentage * PBWIDTH);
    int rpad = PBWIDTH - lpad;
    if (val == 100)
    {
		printf("\r%3d%% [%.*s%*s] done!", val, lpad, PBSTR, rpad, "");
	}
    else
    {
        printf("\r%3d%% [%.*s%*s] recording...", val, lpad, PBSTR, rpad, "");
    }
    fflush(stdout);
}

hipError_t countWithCuda(int* trace, const unsigned int attackLength, const unsigned int traceLength, const unsigned int P);

__global__ void counterKernel(int* trace, const unsigned int attackLength, const unsigned int P, bool *work, int offset)
{
    int counter = 0;
    while (*work)
    {
        counter += 1;
    }
    trace[offset] = counter;
}

int main()
{
    const unsigned int attackLength = 5;
    const unsigned int traceLength = 1000;
    const unsigned int P = 5;
    int trace[traceLength] = { 0 };


    hipError_t cudaStatus = countWithCuda(trace, attackLength, traceLength, P);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "countWithCuda failed!");
        return 1;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    fprintf(stdout, "Trace: ");
    for (int i = 0; i < traceLength; i++)
    {
        fprintf(stdout, "%d ", trace[i]);
    }

    return 0;
}

hipError_t countWithCuda(int* trace, const unsigned int attackLength, const unsigned int traceLength, const unsigned int P)
{
    int offset = 0;
    int *devTrace = 0;
    bool *devWork = 0;
    bool *hostWork = (bool*)malloc(sizeof(bool));
    *hostWork = true;
    hipError_t cudaStatus;
    hipStream_t execStream, transferStream;

    cudaStatus = hipStreamCreate(&execStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed for execStream!");
        goto Error;
    }

    cudaStatus = hipStreamCreate(&transferStream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed for transferStream!");
        goto Error;
    }

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&devTrace, attackLength * 1000 / P * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&devWork, sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(devTrace, trace, traceLength * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(devWork, hostWork, sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "counterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    LARGE_INTEGER frequency;
    LARGE_INTEGER start, stop;

    QueryPerformanceFrequency(&frequency); // Get timer frequency
    QueryPerformanceCounter(&start); // Start timer

    unsigned int hostTime = (int)1000 * ((float)start.QuadPart / frequency.QuadPart);
    const unsigned int attackTimeout = hostTime;
    double elapsed = 0;

    while (attackLength * 1000 >= hostTime - attackTimeout)
    {
        QueryPerformanceCounter(&stop);
        hostTime = (int)1000 * ((float)stop.QuadPart / frequency.QuadPart);
        const unsigned int traceTimeout = hostTime - attackTimeout;

        elapsed = (float)(stop.QuadPart - start.QuadPart) / frequency.QuadPart;

        printProgress(elapsed / attackLength);

        // Launch a kernel on the GPU.
        counterKernel <<<1, 1, 0, execStream>>> (devTrace, attackLength, P, devWork, offset);

        while (P >= hostTime - traceTimeout) // Spin until timeout
        {
            QueryPerformanceCounter(&stop);
			hostTime = (int)1000 * ((float)stop.QuadPart / frequency.QuadPart);
        }

        *hostWork = false;

        cudaStatus = hipMemcpyAsync(devWork, hostWork, sizeof(bool), hipMemcpyHostToDevice, transferStream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching counterKernal!\n", cudaStatus);
            goto Error;
        }

        offset++;

        *hostWork = true;

        cudaStatus = hipMemcpyAsync(devWork, hostWork, sizeof(bool), hipMemcpyHostToDevice, transferStream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching counterKernal!\n", cudaStatus);
            goto Error;
        }
    }

    elapsed = (float)(stop.QuadPart - start.QuadPart) / frequency.QuadPart;

    fprintf(stdout, " Total Time Elapsed: %f\n", elapsed);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(trace, devTrace, traceLength * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(devTrace);
    hipFree(devWork);
    hipStreamDestroy(execStream);
    hipStreamDestroy(transferStream);
    free(hostWork);

    return cudaStatus;
}